#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <>
#include <assert.h>
#include <chrono>
#include <hip/hip_cooperative_groups.h>

#include "MatMul.h"
#include "GPUInfo.h"

typedef struct MatrixD {
    int width;
    int height;
    float* elements;

public:
    __host__ __device__ float& operator[](int index)
    {
        return elements[index];
    }

    __host__ __device__ float operator[](int index) const
    {
        return elements[index];
    }
};

__global__ void MatMulGPUKernel(
    const MatrixD A,
    const MatrixD B,
    MatrixD C
);

__global__ void MatMulGPUKernelWithSharedMemory(
    MatrixD A,
    MatrixD B,
    MatrixD C,
    size_t sizePerBlock,
    size_t blockWidth
);

void MatMulCPU(const Matrix& A, const Matrix& B, Matrix& C)
{
    assert(A.GetWidth() == B.GetHeight(), "A.width should be equal to B.height!");
    auto startTime = std::chrono::system_clock::now();
    for (size_t row = 0; row < A.GetHeight(); row++)
    {
        for (size_t col = 0; col < B.GetWidth(); col++)
        {
            float cvalue = 0;
            for (size_t i = 0; i < B.GetHeight(); i++)
            {
                cvalue += A[row * A.GetWidth() + i] * B[i * B.GetWidth() + col];
            }
            C[row * C.GetWidth() + col] = cvalue;
        }
    }
    auto endTime = std::chrono::system_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime);
    std::cout << duration.count() << "ms" << std::endl;
}

void MatMulGPU(Matrix& A, Matrix& B, Matrix& C)
{
    // CPU -> GPU Memory allocate
    A.ToDevice();
    B.ToDevice();
    C.ToDevice();
    MatrixD d_A;
    MatrixD d_B;
    MatrixD d_C;

    d_A.width = A.GetWidth();
    d_A.height = A.GetHeight();
    d_A.elements = A.GetElements();

    d_B.width = B.GetWidth();
    d_B.height = B.GetHeight();
    d_B.elements = B.GetElements();

    d_C.width = C.GetWidth();
    d_C.height = C.GetHeight();
    d_C.elements = C.GetElements();

    // Load GPU info
    GPUInfo info = GPUInfo(false);
    const int deviceCount = info.GetDeviceCount();
    const hipDeviceProp_t* deviceProps = info.GetDeviceProps();

    // Invoke kernel
    size_t outputSize = C.GetHeight() * C.GetWidth();
    size_t numThreads = std::min((int)outputSize, deviceProps[0].maxThreadsPerBlock);
    dim3 dimBlock(numThreads);
    size_t gridSize = std::ceil((float)outputSize / numThreads);
    gridSize = std::min((int)gridSize, deviceProps[0].maxGridSize[0]);
    dim3 dimGrid(gridSize);
    std::cout << "outputSize: " << outputSize << ", numThreads: " << numThreads << ", gridSize: " << gridSize << std::endl;

    auto startTime = std::chrono::system_clock::now();
    MatMulGPUKernel<<<dimGrid, dimBlock>>>(
        d_A,
        d_B,
        d_C
    );

    hipDeviceSynchronize();
    auto endTime = std::chrono::system_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime);
    std::cout << duration.count() << "ms" << std::endl;

    // GPU -> CPU Memory allocate
    A.ToHost();
    B.ToHost();
    C.ToHost();
}

void MatMulGPUWithSharedMemory(Matrix& A, Matrix& B, Matrix& C)
{
    // CPU -> GPU Memory allocate
    A.ToDevice();
    B.ToDevice();
    C.ToDevice();
    MatrixD d_A;
    MatrixD d_B;
    MatrixD d_C;

    d_A.width = A.GetWidth();
    d_A.height = A.GetHeight();
    d_A.elements = A.GetElements();

    d_B.width = B.GetWidth();
    d_B.height = B.GetHeight();
    d_B.elements = B.GetElements();

    d_C.width = C.GetWidth();
    d_C.height = C.GetHeight();
    d_C.elements = C.GetElements();

    // Load GPU info
    GPUInfo info = GPUInfo(false);
    const int deviceCount = info.GetDeviceCount();
    const hipDeviceProp_t* deviceProps = info.GetDeviceProps();

    // Invoke kernel
    size_t outputSize = d_C.height * d_C.width;
    size_t sharedMemSize = deviceProps[0].sharedMemPerBlock / sizeof(float);
    size_t maxBlockSize = std::floor(std::sqrt(std::min((size_t)deviceProps[0].maxThreadsPerBlock, sharedMemSize / 2)));
    size_t blockSize = std::_Gcd(d_C.height, d_C.width);
    
    if (blockSize > maxBlockSize)
    {
        for (size_t i = maxBlockSize; i > 0; i--)
        {
            if (blockSize % i == 0)
            {
                blockSize = i;
                break;
            }
        }
    }

    size_t numThreads = blockSize * blockSize;
    dim3 dimBlock(numThreads);
    size_t gridSize = std::ceil((float)outputSize / numThreads);
    gridSize = std::min((int)gridSize, deviceProps[0].maxGridSize[0]);
    dim3 dimGrid(gridSize);
    size_t blockWidth = std::ceil((float)d_C.width / blockSize);

    std::cout << "outputSize: " << outputSize << ", numThreads: " << numThreads << ", gridSize: " << gridSize << ", blockSize: " << blockSize << std::endl;

    auto startTime = std::chrono::system_clock::now();
    MatMulGPUKernelWithSharedMemory<<<dimGrid, dimBlock, numThreads * 2 * sizeof(float)>>>(
        d_A,
        d_B,
        d_C,
        blockSize,
        blockWidth
    );

    hipDeviceSynchronize();
    auto endTime = std::chrono::system_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime);
    std::cout << duration.count() << "ms" << std::endl;

    // GPU -> CPU Memory allocate
    A.ToHost();
    B.ToHost();
    C.ToHost();
}

__global__ void MatMulGPUKernel(
    const MatrixD A,
    const MatrixD B,
    MatrixD C
)
{
	float CValue = 0;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= C.height * C.width) 
        return;
    int row = idx / C.width;
    int col = idx % C.width;

	for (int e = 0; e < A.width; ++e)
	{
		CValue += A[row * A.width + e] * B[e * B.width + col];
	}
	C[row * C.width + col] = CValue;
}

__global__ void MatMulGPUKernelWithSharedMemory(
    MatrixD A,
    MatrixD B,
    MatrixD C,
    size_t blockSize,
    size_t blockWidth
)
{
    extern __shared__ float subsetMatrix[];
    // Shared memory used to store Asub and Bsub respectively
    float* As = (float*)subsetMatrix;
    float* Bs = (float*)(subsetMatrix + blockSize * blockSize);

    float CValue = 0;
    int blockRow = blockIdx.x / blockWidth;
    int blockCol = blockIdx.x % blockWidth;
    int row = threadIdx.x / blockSize;
    int col = threadIdx.x % blockSize;
    
    int CSubIdx = (blockRow * C.width + blockCol) * blockSize;
    int curAWidth = A.width;
    int m = 0;
    while (curAWidth > 0)
    {
        blockSize = (curAWidth > blockSize) ? blockSize : curAWidth;

        int ASubIdx = (blockRow * A.width + m) * blockSize;
        int BSubIdx = (m * B.width + blockCol) * blockSize;

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row * blockSize + col] = A[ASubIdx + row * A.width + col];
        Bs[row * blockSize + col] = B[BSubIdx + row * B.width + col];

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < blockSize; ++e)
        {
            CValue += As[row * blockSize + e] * Bs[e * blockSize + col];
        }
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
        curAWidth -= blockSize;
        m++;
    }
    C[CSubIdx + row * C.width + col] = CValue;
}

void MatMulTest()
{
    Matrix A(1000, 1000, true);
    Matrix B(1000, 1000, true);
    Matrix C(1000, 1000, true);
    Matrix D(1000, 1000, true);
    Matrix E(1000, 1000, true);

    FillRandn(A);
    FillRandn(B);
    FillZeros(C);
    FillZeros(D);
    FillZeros(E);

    MatMulCPU(A, B, C);
    MatMulGPU(A, B, D);
    MatMulGPUWithSharedMemory(A, B, E);

    // std::cout << (C == D) << std::endl;
    std::cout << (D == E) << std::endl;

    return;
}